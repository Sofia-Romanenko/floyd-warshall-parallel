#include <iostream>
#include <hip/hip_runtime.h>
#include <limits>

#define INF 1e9

__global__ void floyd_warshall_kernel(int *dist, int n, int k) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n) {
        int ik = dist[i * n + k];
        int kj = dist[k * n + j];
        if (ik < INF && kj < INF) {
            atomicMin(&dist[i * n + j], ik + kj);
        }
    }
}

void floyd_warshall_cuda(int *h_dist, int n) {
    int *d_dist;
    size_t size = n * n * sizeof(int);

    // Copy data to GPU
    hipMalloc(&d_dist, size);
    hipMemcpy(d_dist, h_dist, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((n + 15) / 16, (n + 15) / 16);

    for (int k = 0; k < n; ++k) {
        floyd_warshall_kernel<<<numBlocks, threadsPerBlock>>>(d_dist, n, k);
        hipDeviceSynchronize();
    }

    // Copy result back to CPU
    hipMemcpy(h_dist, d_dist, size, hipMemcpyDeviceToHost);
    hipFree(d_dist);
}

int main() {
    int n = 500;
    int *dist = new int[n * n];

    // Initialize graph (example: random graph)
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
            dist[i * n + j] = (i == j) ? 0 : (rand() % 100 + 1);
        }
    }

    floyd_warshall_cuda(dist, n);

    delete[] dist;
    return 0;
}
